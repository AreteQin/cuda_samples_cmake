// includes, system
#include <stdio.h>
#include <ctime>
#include <iostream>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void increment_kernel(int *g_data, int inc_value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] = g_data[idx] + inc_value;
}

bool correct_output(int *data, const int n, const int x) {
    for (int i = 0; i < n; i++)
        if (data[i] != x) {
            printf("Error! data[%d] = %d, ref = %d\n", i, data[i], x);
            return false;
        }

    return true;
}

int main(int argc, char *argv[]) {
    int devID = 0;
    hipDeviceProp_t deviceProps;

    printf("[%s] - Starting...\n", argv[0]);

    // get device name
    hipGetDeviceProperties(&deviceProps, devID);
    printf("CUDA device [%s]\n", deviceProps.name);

    int n = 16 * 1024 * 1024;
    int nbytes = n * sizeof(int);
    int value = 26;

    // allocate host memory
    int *a = 0;
    hipHostMalloc((void **)&a, nbytes, hipHostMallocDefault);
    memset(a, 0, nbytes);

    // allocate device memory
    int *d_a = 0;
    hipMalloc((void **)&d_a, nbytes);
    hipMemset(d_a, 255, nbytes);

    // set kernel launch configuration
    dim3 threads = dim3(512, 1);
    dim3 blocks = dim3(n / threads.x, 1);

    // create cuda event handles
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipDeviceSynchronize();
    float gpu_time = 0.0f;

    // asynchronously issue work to the GPU (all to stream 0)
    hipProfilerStart();
    // record start time
    clock_t start_time = clock();
    hipEventRecord(start, 0);
    hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);
    increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
    hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
    hipEventRecord(stop, 0);
    // record end time
    clock_t end_time = clock();
    hipProfilerStop();

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter = 0;

    while (hipEventQuery(stop) == hipErrorNotReady) {
        counter++;
    }

    hipEventElapsedTime(&gpu_time, start, stop);

    // print the cpu and gpu times
    printf("time spent executing by the GPU: %.2f\n", gpu_time);
    std::cout << "time spent by CPU in CUDA calls: " << ((float)(end_time - start_time)) / CLOCKS_PER_SEC << std::endl;
    printf("CPU executed %lu iterations while waiting for GPU to finish\n",
           counter);

    // check the output for correctness
    bool bFinalResults = correct_output(a, n, value);

    // release resources
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipHostFree(a);
    hipFree(d_a);

    exit(bFinalResults ? EXIT_SUCCESS : EXIT_FAILURE);
}
